#include "hip/hip_runtime.h"
#include <HW2/Fractal/julia_fractal.cuh>
#include <hip/hip_runtime_api.h>
#include <cmath>

const double R = 2.0;
const int THREAD_SIZE = 16;
const int FRAME_SIZE = 4;
const int MAX_ITER = 256;

namespace CudaStudy{
    __global__ void getFractal(unsigned char* resVid, int height, int width, int frames){
        int frame = blockDim.z * blockIdx.z + threadIdx.z;
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        int y = blockDim.y * blockIdx.y + threadIdx.y;
        if(frame>=frames or x>=height or y>=width) return;
        double zx = 4*(static_cast<double>(x) / height)-2;
        double zy = 4*(static_cast<double>(y) / width)-2;
        double frame_to_pi = 2 * M_PI * ((double)frame/frames);
        double cx = 0.7885 * sin(frame_to_pi);
        double cy = 0.7885 * cos(frame_to_pi);
        int iteration = 0;
        while(iteration < MAX_ITER and zx * zx + zy * zy < R*R){
            double xtemp = zx * zx - zy * zy;
            zy = 2 * zx * zy + cy;
            zx = xtemp + cx;
            iteration++;
        }
        int offset = frame*(width*height) + (x*width + y);
        resVid[offset] = (unsigned char)(iteration);
    }

    void juliaFractal(unsigned char *resVid, int height, int width, int frames){
        unsigned char *cudaVid;
        hipMalloc((void**)&cudaVid, width * height * frames * sizeof(unsigned char));
        dim3 grid(ceil((double)height/THREAD_SIZE), ceil((double)width/THREAD_SIZE), ceil((double)frames/FRAME_SIZE)), thread(THREAD_SIZE, THREAD_SIZE, FRAME_SIZE);
        getFractal<<<grid, thread>>>(cudaVid, height, width, frames);
        hipMemcpy(resVid, cudaVid, sizeof(unsigned char) * height * width * frames, hipMemcpyDeviceToHost);
        hipFree(cudaVid);
    }
}