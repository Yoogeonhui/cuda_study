#include "hip/hip_runtime.h"
#include <HW2/Grey/color_to_greyscale.cuh>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

const int THREAD_SIZE = 16;

namespace CudaStudy{
    __global__ void cudaConvertToGrey(const unsigned char *h_cuda_origImg, unsigned char *h_cuda_resultImg, int height, int width){
        int h = blockDim.x * blockIdx.x + threadIdx.x;
        int w = blockDim.y * blockIdx.y + threadIdx.y;
        
        if(h>=height or w>=width) return;
        h_cuda_resultImg[h * width+w] = static_cast<unsigned char>(h_cuda_origImg[(h*width+w)*3] * 0.114 + h_cuda_origImg[(h*width+w)*3+1] * 0.587 + h_cuda_origImg[(h*width+w)*3+2] * 0.299);
        
    }

    void convertToGrey(const unsigned char *h_origImg, unsigned char *h_resultImg, int height, int width, int channels){
        unsigned char* h_cuda_origImg, *h_cuda_resultImg;
        hipMalloc((void**)&h_cuda_origImg, width*height*channels * sizeof(unsigned char));
        hipMalloc((void**)&h_cuda_resultImg, width*height*sizeof(unsigned char));
        hipMemcpy(h_cuda_origImg, h_origImg, height*width*channels*sizeof(unsigned char), hipMemcpyHostToDevice);

        dim3 grid((double)height/THREAD_SIZE, (double)width/THREAD_SIZE), threads(THREAD_SIZE,THREAD_SIZE);
        cudaConvertToGrey<<<grid, threads>>>(h_cuda_origImg, h_cuda_resultImg, height, width);
        hipDeviceSynchronize();
        hipMemcpy(h_resultImg, h_cuda_resultImg, height*width*sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipFree(h_cuda_origImg);
        hipFree(h_cuda_resultImg);
        return;
    }
}